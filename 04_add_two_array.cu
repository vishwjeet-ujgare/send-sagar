#include "hip/hip_runtime.h"
#include<stdio.h>

#include<hip/hip_runtime.h>

__global__ void add_arrays(int *c , const int  *a, const int *b , int size){
    int i=blockIdx.x*blockDim.x+threadIdx.x;

    if(i<size){
        c[i]=a[i]+b[i];
    }
}


int main(){
    const int size=5;

    int a[size]={1,2,3,4,5};
    int b[size]={1,2,3,4,5};
    int *d_c;

    //Allocate memory on the device for array c
    hipMalloc((void**)&d_c,size * sizeof(int));

    //Copy arrays a and b to the device
    int *d_a,*d_b;

    hipMalloc((void**)&d_a , size*sizeof(int));
    hipMalloc((void**)&d_b , size*sizeof(int));
    
    hipMemcpy(d_a,a,size * sizeof(int),cudaMemcpyHostoDevice);
    hipMemcpy(d_b,a,size * sizeof(int),cudaMemcpyHostoDevice);


    add_arrays<<<2,4>>>(d_c,d_a,d_b,size);
    hipDeviceSynchronize();

    //copy thre result back from the device

    int *c =(int*)malloc(5 * sizeof(int));
    hipMemcpy(c,d_c,size*sizeof(int),hipMemcpyDeviceToHost);

    //print the reslult
    for (int i=0;i<size;i++){
        printf("%d ",c[i]);
    }
    printf("\n");
    //free me
    hipFree(c);
    hipFree(d_a);
    hipFree(d_c);
    hipFree(d_b);


}