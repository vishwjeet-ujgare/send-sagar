#include "hip/hip_runtime.h"

#include <stdio.h>


//CUDA kernel to add two integers
__global__ void addIntegers(int* a ,  int* b , int* result){
    *result =*a+*b;
    printf("In GPU ... Sum id %d\n",*result);
}

int main(){
    //Host variables
    int host_a=5;
    int host_b=7;
    int host_result=0;

    //Declare for Device variables
    int *device_a;
    int *device_b;
    int *device_result;

    //Allocate memory on the device/host
    hipMalloc((void**)&device_a,sizeof(int));
    hipMalloc((void**)&device_b,sizeof(int));
    hipMalloc((void**)&device_result,sizeof(int));

    //copy data from host to device
    hipMemcpy(device_a,&host_a,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(device_b,&host_b,sizeof(int),hipMemcpyHostToDevice);

    //launch the kernel with one block and one head
    
    // addIntegers <<<1,1>>>(device_a,device_b,device_result);
    // addIntegers <<<1,5>>>(device_a,device_b,device_result);
    // addIntegers <<<5,5>>>(device_a,device_b,device_result);
    // addIntegers <<<1,1025>>>(device_a,device_b,device_result);

    //1 block containers at most 1024
    addIntegers <<<1,1024>>>(device_a,device_b,device_result);

    //copy the result from device to host
    hipMemcpy(&host_result,device_result,sizeof(int),hipMemcpyDeviceToHost);

    //Display the result
    printf("Sum of %d and %d is %d\n",host_a,host_b, host_result);

    //free alloacted memory
    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_result);
   
    return 0;



}